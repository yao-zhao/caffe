#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/sinh_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SinHForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = sinh(in[index]);
  }
}

template <typename Dtype>
void SinHLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SinHForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void SinHBackward(const int n, const Dtype* top_diff,
    const Dtype* bottom_data, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    bottom_diff[index] = cosh(bottom_data[index])*top_diff[index];
  }
}

template <typename Dtype>
void SinHLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SinHBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SinHLayer);


}  // namespace caffe
